#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"



__global__ void calculate_ed(const float * const Subject,const float * const Query, const long int M, const int N,float * gposbitmap,float bsf) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float dist=0;
	long int poss;
	float currentbsfdist=bsf;
	for (int i = thid; i < M; i += gridDim.x*blockDim.x)
	{	
		dist=0;
		poss=i*N;
		for(int j =0;j<N;j++)
		{
			dist += (Subject[poss+j]-Query[j])*(Subject[poss+j]-Query[j]);
			if(dist>currentbsfdist)
			{
				break;
			}
		}
		gposbitmap[i]=dist;
	}
}

__global__ void collectbsf(float *d_data, int N)
{
	int tid = threadIdx.x;
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	float *data = d_data + blockIdx.x*blockDim.x;
	if (index >= N) return;
	for (int strize = 1; strize < blockDim.x; strize *= 2)
	{
		int idx = tid*strize * 2;
		if (idx < blockDim.x)
		{
			if(data[idx]> data[idx+strize])
			data[idx]= data[idx+strize];
		}
		__syncthreads();
	}
}



int main(int argc, char **argv)
{
// hello from cpu

    hipSetDevice(0); 
	float *gposbitmap = NULL;
	float *posbitmap = NULL;
	float *qts = NULL,*gqts = NULL;
	float *dataset=NULL,*gdataset=NULL;

//helloFromGPU <<<1, 10>>>();
	struct timeval total_time_start;
	struct timeval current_time;
	double total_time;
	double tS;
	double tE;

	FILE * dfile;
	dfile = fopen (argv[1],"rb");
	FILE * qfile;
	qfile = fopen (argv[2],"rb");

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	unsigned long datasize = atoi(argv[3]);
	int queriessize = atoi(argv[4]);
	int timeseriessize = atoi(argv[5]);
	unsigned long chunksize=atoi(argv[6]);
	int i;

	//i//nt breakpointsize=sizeof(sax_breakpoints);

	hipHostMalloc(&qts, sizeof(float)*timeseriessize, hipHostMallocDefault);
	hipHostMalloc(&dataset, sizeof(float)*datasize*timeseriessize, hipHostMallocDefault); 
	hipHostMalloc(&posbitmap, sizeof(float)*datasize, hipHostMallocDefault); 
	hipMalloc(&gposbitmap, sizeof(float)*datasize); 
	hipMalloc(&gdataset, sizeof(float)*chunksize*timeseriessize); 
	hipMalloc(&gqts, sizeof(float)*timeseriessize); 
	fread(dataset, sizeof(float),timeseriessize*datasize,dfile);
	gettimeofday(&total_time_start, NULL);   
	hipEventRecord(start, 0);
	printf("finished to read data\n");
	for(i=0; i<queriessize;i++)
	{
		float BSF=2000000.0f;
		fread(qts, sizeof(float),timeseriessize,qfile);
		hipMemcpy(gqts, qts, sizeof(float)*timeseriessize,hipMemcpyHostToDevice);

		

		unsigned long int currentchunksize;
		for(unsigned long j=0;j<datasize;j+=chunksize)
		{
			if(j+chunksize<=datasize)
			{
				currentchunksize=chunksize;
			}
			else
			{
				currentchunksize=j-datasize;
			}
			int streamnumber=10;
			hipStream_t streams[streamnumber];
			for(int k=0;k<streamnumber;k++)
			{
				hipStreamCreate(&streams[k]);
			}
			for( int k=0;k<streamnumber;k++)
			{
				hipMemcpyAsync(&gdataset[k*timeseriessize*currentchunksize/streamnumber], &dataset[j*timeseriessize+k*timeseriessize*currentchunksize/streamnumber], sizeof(float)*timeseriessize*currentchunksize/streamnumber,hipMemcpyHostToDevice,streams[i]);
				calculate_ed<<<500,200,10,streams[i]>>> (&gdataset[k*timeseriessize*currentchunksize/streamnumber], gqts, currentchunksize/streamnumber, timeseriessize,&gposbitmap[j+k*currentchunksize/streamnumber],BSF); 
				hipMemcpyAsync(&posbitmap[j+k*currentchunksize/streamnumber], &gposbitmap[j+k*currentchunksize/streamnumber], sizeof(float)*currentchunksize/streamnumber,hipMemcpyDeviceToHost,streams[i]);
			}
			hipDeviceSynchronize();
		}
		for(unsigned long j=0;j<datasize;j++)
		{
			if(posbitmap[j]<BSF)
			BSF=posbitmap[j];
		}
		printf("the distance is %f \n",BSF);
	}
	hipError_t aaaa= hipDeviceSynchronize();
	hipDeviceSynchronize();
	gettimeofday(&current_time, NULL);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
            
	tS = total_time_start.tv_sec*1000000 + (total_time_start.tv_usec); 
	tE = current_time.tv_sec*1000000  + (current_time.tv_usec); 
	total_time = (tE - tS); 
	//printf("The distance is %ld\n",sizeof(float));
	printf("the time is \t%f\n",total_time);
	hipDeviceReset();

	return 0;
}
