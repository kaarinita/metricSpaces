#include "hip/hip_runtime.h"
#include <float.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "singlib.h"



__global__ void calculate_lbd(const sax_type * const saxarray,const float * const paa, const long int M, const int N,bool * positionarray,const float BSF,float segmentsize) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	float lbsf=BSF/segmentsize;

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -

	
	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) {
                	if(distance<lbsf)
		{
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
				float breaku=((float)region_lower-127.0f)/128.0f;
            			breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
				if(breakpoint_upper < paa[i])
				{
            				distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        			}
			}
        		else if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
				if (breakpoint_lower > paa[i]) 
				{
            				distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        			}
        		}
        		else 
			{
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
				if (breakpoint_lower > paa[i]) 
				{
            				distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        			}
				else
				{
					float breaku=((float)region_lower-127.0f)/128.0f;
            				breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
					if(breakpoint_upper < paa[i])
					{
            					distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        				}
        			} 

        		}
		}
						        		
    	}

		if(distance<lbsf)
		{positionarray[j]=true;}
		else
		{positionarray[j]=false;}
	}
}



__global__ void calculate_lbdfloat(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * positionarray,const float BSF, float segmentsize) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) 
		{
			if(segmentsize*distance<BSF)
			{
				sax_type v = saxarray[j*N+i];
				sax_type region_lower = v ;//shift operation 
				sax_type region_upper = (~((int)MAXFLOAT) | region_lower);
				if (region_lower == 0)
				{
					breakpoint_lower = -2000000;
					float breaku=((float)region_lower-127.0f)/128.0f;
					breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
					if(breakpoint_upper < paa[i])
					{
						distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
					}
				}
				else if (region_upper == 256 - 1)
				{
					breakpoint_upper = +2000000;
					float breakx=((float)region_lower-128.0f)/128.0f;
					breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
					if (breakpoint_lower > paa[i]) 
					{
						distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
					}
				}
				else
				{
					float breakx=((float)region_lower-128.0f)/128.0f;
					breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
					if (breakpoint_lower > paa[i]) 
					{
						distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
					}
					else
					{
						float breaku=((float)region_lower-127.0f)/128.0f;
						breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
						if(breakpoint_upper < paa[i])
						{
							distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
						}
					}
				}
			}
		}
		positionarray[j]=segmentsize*distance;
	}
}









extern "C" float* initialGPU(float *qts, float *gqts, sax_type *saxarray, sax_type *gsaxarray, float *dictionary, float *gdictionary,unsigned long datasize,float *sax_breakpoints )
{

	hipSetDevice(0); 
	hipMalloc(&gqts, sizeof(float)*16); 
	return gqts;
}
extern "C" float* initialgqts(float *gqts)
{
		hipMalloc(&gqts, sizeof(float)*16); 
		return gqts;
}
extern "C" void GPUsyn()
{
hipDeviceSynchronize();
}




extern "C" float* initialgdictionary(float *gdictionary)
{
		hipMalloc(&gdictionary, sizeof(float)*257);

		return gdictionary;
}
extern "C" bool* initialgposbitmap(bool *gposbitmap,unsigned long datasize)
{
	hipMalloc(&gposbitmap, sizeof(bool)*datasize); 
		return gposbitmap;
}
extern "C" bool* initialposbitmap(bool *posbitmap,unsigned long datasize)
{
	hipHostMalloc(&posbitmap, sizeof(bool)*datasize); 
		return posbitmap;
}
extern "C" float* initialgposbitmapfloat(float *gposbitmap,unsigned long datasize)
{
	hipMalloc(&gposbitmap, sizeof(float)*datasize); 
		return gposbitmap;
}
extern "C" float* initialposbitmapfloat(float *posbitmap,unsigned long datasize)
{
hipHostMalloc(&posbitmap, sizeof(float)*datasize); 
		return posbitmap;
}

extern "C" sax_type* initialgsaxarray(sax_type *gsaxarray,unsigned long datasize)
{
	hipMalloc(&gsaxarray, sizeof(sax_type)*datasize*16); 

		return gsaxarray;
}
extern "C" sax_type* initialsaxarray(sax_type *saxarray,unsigned long datasize)
{
	hipHostMalloc(&saxarray, sizeof(sax_type)*datasize*16); 

		return saxarray;
}
extern "C" void initialdevice()
{
	hipSetDevice(0);
}
extern "C" void gpumemcpy(sax_type *gsaxarray,sax_type *saxarray,unsigned long datasize)
{	

	hipMemcpy(gsaxarray, saxarray,sizeof(sax_type)*datasize*16,hipMemcpyHostToDevice);
}
extern "C" void gpusaxgridmemcpy(sax_type *gsaxarray,sax_type *saxarray,unsigned long datasize)
{	
	hipMemcpy(gsaxarray, saxarray,sizeof(sax_type)*datasize*16,hipMemcpyHostToDevice);
}

extern "C" void gpudictionarymemcpy(float *gdictionary,float *sax_breakpoints)
{	
int offset = ((256 - 1) * (256 - 2)) / 2;
	hipMemcpy(gdictionary, &sax_breakpoints[offset-1], sizeof(float)*257,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}





extern "C" void GPUfree(void *devicememorypointer)
{
	hipFree(devicememorypointer);
}






extern "C" void LBDfloatstreamGPU(sax_type *saxarray, float *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,float * gposbitmap,int segmentnumber,float segmentsize)
{
	int streamnumber=10;
	hipMemcpy(gqts, qts,sizeof(float)*segmentnumber,hipMemcpyHostToDevice);
	hipStream_t streams[streamnumber];
	for(int i=0;i<streamnumber;i++)
	{
		hipStreamCreate(&streams[i]);
	}

	for(int i=0;i<streamnumber;i++)
	{
		calculate_lbdfloat<<<200,500,10,streams[i]>>> (saxarray+i*datasize*segmentnumber/streamnumber,gqts, datasize/streamnumber, segmentnumber, gposbitmap+i*datasize/streamnumber,BSF,segmentsize); 
        hipMemcpyAsync(posbitmap+i*datasize/streamnumber, gposbitmap+i*datasize/streamnumber, sizeof(float)*datasize/streamnumber,hipMemcpyDeviceToHost,streams[i]);
	}
	hipDeviceSynchronize();
}




extern "C" void LBDstreamGPU(sax_type *saxarray, bool *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool * gposbitmap,int segmentnumber,float segmentsize)
{
	int streamnumber=20;
	hipMemcpy(gqts, qts,sizeof(float)*segmentnumber,hipMemcpyHostToDevice);
	hipStream_t streams[streamnumber];
	for(int i=0;i<streamnumber;i++)
	{
		hipStreamCreate(&streams[i]);
	}
	for(int i=0;i<streamnumber;i++)
	{
		calculate_lbd<<<200,500,10,streams[i]>>> (saxarray+i*datasize*segmentnumber/streamnumber,gqts, datasize/streamnumber, segmentnumber,gposbitmap+i*datasize/streamnumber,BSF, segmentsize); 
		hipMemcpyAsync(posbitmap+i*datasize/streamnumber, gposbitmap+i*datasize/streamnumber, sizeof(bool)*datasize/streamnumber,hipMemcpyDeviceToHost,streams[i]);
	}
	hipDeviceSynchronize();
}





















/*
extern "C" void SIMSlowerGPU(sax_type *saxarray, bool *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool * gposbitmap,float *gdictionary)
{

	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	calculate_lbd<<<200,200,10>>> (saxarray,gqts, datasize, 16, gposbitmap,BSF,16); 

	hipMemcpy(posbitmap, gposbitmap, sizeof(bool)*datasize,hipMemcpyDeviceToHost);
//hipDeviceSynchronize();


}
extern "C" void SIMSlowerGPUfloat(sax_type *saxarray, float *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,float * gposbitmap,float *gdictionary)
{

	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	calculate_lbdfloat<<<200,500,10>>> (saxarray,gqts, datasize, 16,gdictionary, gposbitmap,BSF,16); 
	hipMemcpy(posbitmap, gposbitmap, sizeof(float)*datasize,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
}

extern "C" void initialGPU_grid2(float *qts, float *gqts, sax_type **saxarray, sax_type **gsaxarray, bool **posbitmap, bool **gposbitmap, float *dictionary, float *gdictionary, unsigned long datasize,unsigned long *gridsize,float *sax_breakpoints )
{
	hipSetDevice(0); 

	for(int i =0;i<65536;i++)
	{
		hipHostMalloc(&saxarray[i], sizeof(sax_type)*gridsize[i]*16); 
		hipMalloc(&gposbitmap[i], sizeof(float)*gridsize[i]); 
		hipMalloc(&gsaxarray[i], sizeof(float)*gridsize[i]*16); 
		hipHostMalloc(&posbitmap[i], sizeof(float)*gridsize[i]); 
		hipMemcpy(gsaxarray[i], saxarray[i],sizeof(float)*gridsize[i]*16,hipMemcpyHostToDevice);
	}
	hipHostMalloc(&dictionary, 257*sizeof(float)); 
	hipHostMalloc(&qts, sizeof(float)*256); 
	int offset = ((256 - 1) * (256 - 2)) / 2;
	//memcpy(dictionary,&sax_breakpoints[offset-1],sizeof(float)*257);

	hipMalloc(&gdictionary, sizeof(float)*257);
	hipMalloc(&gqts, sizeof(float)*256); 

	hipMemcpy(gdictionary, &sax_breakpoints[offset-1], sizeof(float)*257,hipMemcpyHostToDevice);
}
extern "C" void SIMSlowerGPUgridstream(sax_type **saxarray, bool **posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool ** gposbitmap,float *gdictionary,long int *nodemap, unsigned long *gridnumber)
{
	int streamnumber=(int)datasize;
	hipStream_t streams[streamnumber];
	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	for(int i=0;i<streamnumber;i++)
	{
		hipStreamCreate(&streams[i]);
	}

	for(long int i=0;i<streamnumber;i++)
	{

		calculate_lbd2<<<500,100,10,streams[i]>>> (saxarray[nodemap[i]],gqts, gridnumber[nodemap[i]], 16,gdictionary, gposbitmap[nodemap[i]],BSF); 
        hipMemcpyAsync(posbitmap[nodemap[i]], gposbitmap[nodemap[i]], sizeof(bool)*gridnumber[nodemap[i]],hipMemcpyDeviceToHost,streams[i]);
	}
	for(long int i=0;i<datasize;i++)
	{
		//hipMemcpy(posbitmap[nodemap[i]], gposbitmap[nodemap[i]], sizeof(bool)*gridnumber[nodemap[i]],hipMemcpyDeviceToHost,streams[i]);
	}
	hipDeviceSynchronize();
}
extern "C" void SIMSstreamlowerGPU2(sax_type *saxarray, bool *posbitmap,float * qts,float * gqts,float BSF,unsigned long int startnumber,unsigned long int stopnomber,bool * gposbitmap,float *gdictionary)
{

	int streamnumber=10;
	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	unsigned long int datasize=100000000;

	int aaa=startnumber/100000000;
	int bbb=stopnomber/100000000+1;

	if(bbb>=10)
	bbb=10;


	hipStream_t streams[streamnumber];
	for(int i=0;i<streamnumber;i++)
	{
		hipStreamCreate(&streams[i]);
	}
	for(int i=0;i<streamnumber;i++)
	{
		calculate_lbd3<<<200,500,10,streams[i]>>> (saxarray+i*datasize*16/streamnumber,gqts, datasize/streamnumber, 16,gposbitmap+i*datasize/streamnumber,BSF,16.0); 
       	hipMemcpyAsync(posbitmap+i*datasize/streamnumber, gposbitmap+i*datasize/streamnumber, sizeof(bool)*datasize/streamnumber,hipMemcpyDeviceToHost,streams[i]);
	}

	hipDeviceSynchronize();

}
extern "C" void SIMSlowerGPUgrid(sax_type **saxarray, bool **posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool ** gposbitmap,float *gdictionary,long int *nodemap, unsigned long *gridnumber)
{
	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
	for(long int i=0;i<datasize;i++)
	{

		calculate_lbd2<<<500,200,10>>> (saxarray[nodemap[i]],gqts, gridnumber[nodemap[i]], 16,gdictionary, gposbitmap[nodemap[i]],BSF); 
	}
	for(long int i=0;i<datasize;i++)
	{
		hipMemcpy(posbitmap[nodemap[i]], gposbitmap[nodemap[i]], sizeof(bool)*gridnumber[nodemap[i]],hipMemcpyDeviceToHost);
	}
	hipDeviceSynchronize();
}

__global__ void calculate_lbd4(const sax_type * const saxarray,const float * const paa, const long int M, const int N,bool * positionarray,const float BSF,const long int offset) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	float lbsf=BSF/16.0;

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -

	
	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) {
                	if(distance<lbsf)
		{
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
				float breaku=((float)region_lower-127.0f)/128.0f;
            			breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
				if(breakpoint_upper < paa[i])
				{
            				distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        			}
			}
        		else if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
				if (breakpoint_lower > paa[i]) 
				{
            				distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        			}
        		}
        		else 
			{
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
				if (breakpoint_lower > paa[i]) 
				{
            				distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        			}
				else
				{
					float breaku=((float)region_lower-127.0f)/128.0f;
            				breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
					if(breakpoint_upper < paa[i])
					{
            					distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        				}
        			} 

        		}
		}
						        		
    	}

		if(distance<lbsf)
		{positionarray[j]=true;}
		else
		{positionarray[j]=false;}
	}
}


__global__ void calculate_lbd2float(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * const sax_breakpoints,float * positionarray,const float BSF) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) 
		{
        	if(16*distance<BSF)
		{
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
        		}
        		else
        		{
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
        		}




        		if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
        		}
        		else 
			{
			float breaku=((float)region_lower-127.0f)/128.0f;
            			breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
        		} 

	        			

        		if (breakpoint_lower > paa[i]) 
			{

            			distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        		}
        		else if(breakpoint_upper < paa[i])
			{
            			distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        		}
		}
			        				        		
    		}


		positionarray[j]=16*distance;
	}
}

__global__ void calculate_ed2(const float * const Subject, const float * const Query, const long int M, const int N,float * gposbitmap) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float dist=0;
	long int poss;
	//printf("hello jfeowjfw %d\n",thid);
	for (int i = thid; i < M; i += gridDim.x*blockDim.x)
	{	
		dist=0;
		poss=i*N;
		for(int j =0;j<N;j++)
		{
        		dist += (Subject[poss+j]-Query[j])*(Subject[poss+j]-Query[j]);
				
		}
		gposbitmap[i]=dist;
		//if(dist<BSF)
		//resultmap[i]=dist;
	}		
}
__global__ void calculate_lbdold(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * const sax_breakpoints,bool * positionarray,const float BSF) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) {
        
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
        		}
        		else
        		{
           			breakpoint_lower = sax_breakpoints[region_lower];//(float)(region_lower-128)*(region_lower-128)/16484.0f;//sax_breakpoints[region_lower];
        		}

        		if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
        		}
        		else
        		{
            			breakpoint_upper = sax_breakpoints[region_lower+1];//(float)(region_upper+1-128)*(region_upper+1-128)/16484.0f;//sax_breakpoints[region_upper+1];//search in a list(why?)
        		}

	        			

        		if (breakpoint_lower > paa[i]) 
			{

            			distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        		}
        		else if(breakpoint_upper < paa[i])
			{
            			distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        		}

    		}

		if(16*distance<BSF)
		{positionarray[j]=true;}
		else
		{positionarray[j]=false;}
	}
}
__global__ void calculate_lbd2(const sax_type * const saxarray,const float * const paa, const long int M, const int N,float * const sax_breakpoints,bool * positionarray,const float BSF) 
{
	const int thid = blockDim.x*blockIdx.x + threadIdx.x;
	float distance = 0;
	

	int i=0;
        		float breakpoint_lower = 0; // <-- TODO: calculate breakpoints.
        		float breakpoint_upper = 0; // <-- - || -


	for (int j = thid; j < M; j += gridDim.x*blockDim.x)
	{
		distance=0;
		for (i=0; i<N; i++) {
                	if(16*distance<BSF)
		{
        	
        		sax_type v = saxarray[j*N+i];

        		sax_type region_lower = v ;//shift operation 
        		sax_type region_upper = (~((int)MAXFLOAT) | region_lower);



        	
        		if (region_lower == 0)
			{
	            		breakpoint_lower = -2000000;
        		}
        		else
        		{
				float breakx=((float)region_lower-128.0f)/128.0f;
           			breakpoint_lower = breakx*(breakx*breakx*1.1362582192+0.99800);//sax_breakpoints[region_lower];
        		}

        		if (region_upper == 256 - 1) 
			{
            			breakpoint_upper = +2000000;
        		}
        		else 
			{
			float breaku=((float)region_lower-127.0f)/128.0f;
            			breakpoint_upper = breaku*(1.1362582192*breaku*breaku+0.99800);//sax_breakpoints[region_upper+1];//search in a list(why?)
        		} 

	        			

        		if (breakpoint_lower > paa[i]) 
			{

            			distance += (breakpoint_lower - paa[i])*(breakpoint_lower - paa[i]);
        		}
        		else if(breakpoint_upper < paa[i])
			{
            			distance += (breakpoint_upper - paa[i])*(breakpoint_upper - paa[i]);
        		}
}
						        		
    		}

		if(16*distance<BSF)
		{positionarray[j]=true;}
		else
		{positionarray[j]=false;}
	}
}
extern "C" void SIMSlowerGPUsmall(sax_type *saxarray, bool *posbitmap,float * qts,float * gqts,float BSF,long unsigned datasize,bool * gposbitmap,float *gdictionary)
{

	calculate_lbd<<<20,20,10>>> (saxarray,gqts, datasize, 16,gdictionary, gposbitmap,BSF); 

	hipMemcpy(posbitmap, gposbitmap, sizeof(bool)*datasize,hipMemcpyDeviceToHost);

}
extern "C" void copyqts(float * qts,float * gqts)
{
	hipMemcpy(gqts, qts,sizeof(float)*16,hipMemcpyHostToDevice);
}
*/